#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <math.h>
#include "ImageWriter.h"

//First kernel for populating max and working buf
__global__
void firstKernel(int nRows, int nCols, int nSheets, int pt,
                            unsigned char* gpuArray,
                            unsigned char* maxBuf,
                            float* workingBuf){

    //Define a row and col for particular thread
    //These may not correspond to the voxel image's rows and cols
    int row = threadIdx.x;
    int col = blockIdx.x;

    int max = 0;
    float sum = 0.0;
    int offset;

    //Depending on projection type, iterate through image and for each pixel
    //on face, calculate max and sum of the direction facing either
    //towards or away
    if(pt == 1){
        for(int i = 0; i < nSheets; i++){
            offset = i*nRows*nCols + col*nRows + row;
            if(gpuArray[offset] > max){
                max = gpuArray[offset];
            }
            sum += gpuArray[offset] * (i+1.0)/nSheets;
        }
        maxBuf[row*nCols + col] = max;
        workingBuf[row*nCols + col] = sum;
    }else if(pt == 2){
        for(int i = nSheets - 1; i >= 0; i--){
            offset = i*nRows*nCols + col*nRows + row;
            if(gpuArray[offset] > max){
                max = gpuArray[offset];
            }
            sum += gpuArray[offset] * (i+1.0)/nSheets;
        }
        maxBuf[row*nCols + (nRows - col)] = max;
        workingBuf[row*nCols + (nRows - col)] = sum;
    }else if(pt == 3){
        for(int i = nCols - 1; i >= 0; i--){
            offset = row*nRows*nCols + i*nRows + col;
            if(gpuArray[offset] > max){
                max = gpuArray[offset];
            }
            sum += gpuArray[offset] * (i+1.0)/nCols;
        }
        maxBuf[col*nSheets + row] = max;
        workingBuf[col*nSheets + row] = sum;
    }else if(pt == 4){
        for(int i = 0; i < nCols; i++){
            offset = row*nRows*nCols + i*nRows + col;
            if(gpuArray[offset] > max){
                max = gpuArray[offset];
            }
            sum += gpuArray[offset] * (i+1.0)/nCols;
        }
        maxBuf[col*nSheets + (nSheets - row)] = max;
        workingBuf[col*nSheets + (nSheets - row)] = sum;
    }else if(pt == 5){
        for(int i = nRows - 1; i >= 0; i--){
            offset = col*nRows*nCols + row*nRows + i;
            if(gpuArray[offset] > max){
                max = gpuArray[offset];
            }
            sum += gpuArray[offset] * (i+1.0)/nRows;
        }
        maxBuf[col*nCols + row] = max;
        workingBuf[col*nCols + row] = sum;
    }else if(pt == 6){
        for(int i = 0; i < nRows; i++){
            offset = col*nRows*nCols + row*nRows + i;
            if(gpuArray[offset] > max){
                max = gpuArray[offset];
            }
            sum += gpuArray[offset] * (i+1.0)/nRows;
        }
        maxBuf[(nSheets - col)*nCols + row] = max;
        workingBuf[(nSheets - col)*nCols + row] = sum;
    }
}

//Kernel for taking working buf and weighing the values to make a new sum
__global__
void secondKernel(int nRows, int nCols, int nSheets, int pt, float* workingBuf, int size, unsigned char* sumBuf){
    //Define rows and cols again
    int row = threadIdx.x;
    int col = blockIdx.x;

    //Calculate the max sum in the workingBuf
    float maxWeightedSum = 0;
    for(int i = 0; i < size; i++){
        if(workingBuf[i] > maxWeightedSum){
            maxWeightedSum = workingBuf[i];
        }
    }

    //Depending on projection type, weigh the sum so that it is between 0 and 255
    if(pt == 1){
        sumBuf[row*nCols + col] = round(workingBuf[row*nCols + col]/maxWeightedSum*255.0);
    }else if(pt == 2){
        sumBuf[row*nCols + (nRows - col)] = round(workingBuf[row*nCols + (nRows - col)]/maxWeightedSum*255.0);
    }else if(pt == 3){
        sumBuf[col*nSheets + row] = round(workingBuf[col*nSheets + row]/maxWeightedSum*255.0);
    }else if(pt == 4){
        sumBuf[col*nSheets + (nSheets - row)] = round(workingBuf[col*nSheets + (nSheets - row)]/maxWeightedSum*255.0);
    }else if(pt == 5){
        sumBuf[col*nCols + row] = round(workingBuf[col*nCols + row]/maxWeightedSum*255.0);
    }else if(pt == 6){
        sumBuf[(nSheets - col)*nCols + row] = round(workingBuf[(nSheets - col)*nCols + row]/maxWeightedSum*255.0);
    }
}

void writeTheFile(std::string fName, int xres, int yres, const unsigned char* imageBytes, std::string type){
    //Append type to end of filename
    if(type == "max"){
        fName.append("MAX.png");
    }else if(type == "sum"){
        fName.append("SUM.png");
    }

	unsigned char* row = new unsigned char[3*xres];
	ImageWriter* w = ImageWriter::create(fName,xres,yres);
	int next = 0;
	for (int r=0 ; r<yres ; r++)
	{
		for (int c=0 ; c<3*xres ; c+=3)
		{
			row[c] = row[c+1] = row[c+2] = imageBytes[next++];
		}
		w->addScanLine(row);
	}
	w->closeImageFile();
	delete w;
	delete [] row;
}


int main(int argc, char* argv[]){

    //Get command line arguments
    int nRows = atoi(argv[1]);
    int nCols = atoi(argv[2]);
    int nSheets = atoi(argv[3]);
    int pt = atoi(argv[5]);
    std::string outFileName = argv[6];

    //Allocate host array
    unsigned char *array = new unsigned char[nRows * nCols * nSheets];

    //Open and read file, passing voxel data into host array
    std::ifstream file (argv[4], std::ifstream::binary);
    file.seekg (0, file.end);
    int length = file.tellg();
    file.seekg (0, file.beg);
    file.read(reinterpret_cast<char*>(array), length);
    file.close();

    //Copy the voxel data to the GPU
    size_t size = nRows * nCols * nSheets * sizeof(char);
    unsigned char* gpuArray;
    hipMalloc((void**)&gpuArray, size);
    hipMemcpy(gpuArray, array, size, hipMemcpyHostToDevice);

    //Allocate three GPU buffers based on projection type
    if(pt == 1 || pt == 2){
        size = nRows * nCols * sizeof(char);
    }else if(pt == 3 || pt == 4){
        size = nRows * nSheets * sizeof(char);
    }else if(pt == 5 || pt == 6){
        size = nCols * nSheets * sizeof(char);
    }
    unsigned char* maxBuf;
    unsigned char* sumBuf;
    float* workingBuf;
    hipMalloc((void**)&maxBuf, size);
    hipMalloc((void**)&sumBuf, size);
    hipMalloc((void**)&workingBuf, size);

    //CPU buffers
    unsigned char *h_buf;
    unsigned char *h_sumBuf;

    //Define threads per block, and blocks per grid, then launch kernels
    //Copy GPU buffers back to CPU
    //Write images
    if(pt == 1 || pt == 2){
        dim3 threadsPerBlock(nRows);
        dim3 blocksPerGrid(nCols);
        firstKernel<<<blocksPerGrid, threadsPerBlock>>>(nRows, nCols, nSheets, pt, gpuArray, maxBuf, workingBuf);
        secondKernel<<<blocksPerGrid, threadsPerBlock>>>(nRows, nCols, nSheets, pt, workingBuf, size, sumBuf);
        h_buf = new unsigned char[nRows*nCols];
        h_sumBuf = new unsigned char[nRows*nCols];
        hipMemcpy(h_buf, maxBuf, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_sumBuf, sumBuf, size, hipMemcpyDeviceToHost);
        writeTheFile(outFileName, nCols, nRows, h_buf, "max");
        writeTheFile(outFileName, nCols, nRows, h_sumBuf, "sum");
    }else if(pt == 3 || pt == 4){
        dim3 threadsPerBlock(nSheets);
        dim3 blocksPerGrid(nRows);
        firstKernel<<<blocksPerGrid, threadsPerBlock>>>(nRows, nCols, nSheets, pt, gpuArray, maxBuf, workingBuf);
        secondKernel<<<blocksPerGrid, threadsPerBlock>>>(nRows, nCols, nSheets, pt, workingBuf, size, sumBuf);
        h_buf = new unsigned char[nRows*nSheets];
        h_sumBuf = new unsigned char[nRows*nSheets];
        hipMemcpy(h_buf, maxBuf, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_sumBuf, sumBuf, size, hipMemcpyDeviceToHost);
        writeTheFile(outFileName, nSheets, nRows, h_buf, "max");
        writeTheFile(outFileName, nSheets, nRows, h_sumBuf, "sum");
    }else if(pt == 5 || pt == 6){
        dim3 threadsPerBlock(nCols);
        dim3 blocksPerGrid(nSheets);
        firstKernel<<<blocksPerGrid, threadsPerBlock>>>(nRows, nCols, nSheets, pt, gpuArray, maxBuf, workingBuf);
        secondKernel<<<blocksPerGrid, threadsPerBlock>>>(nRows, nCols, nSheets, pt, workingBuf, size, sumBuf);
        h_buf = new unsigned char[nSheets*nCols];
        h_sumBuf = new unsigned char[nSheets*nCols];
        hipMemcpy(h_buf, maxBuf, size, hipMemcpyDeviceToHost);
        hipMemcpy(h_sumBuf, sumBuf, size, hipMemcpyDeviceToHost);
        writeTheFile(outFileName, nCols, nSheets, h_buf, "max");
        writeTheFile(outFileName, nCols, nSheets, h_sumBuf, "sum");
    }

    //Delete all buffers
    delete array;
    delete h_buf;
    delete h_sumBuf;
    hipFree(gpuArray);
    hipFree(maxBuf);
    hipFree(sumBuf);
    hipFree(workingBuf);
    return 0;
}
